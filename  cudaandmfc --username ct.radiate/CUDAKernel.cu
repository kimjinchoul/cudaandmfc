#include "hip/hip_runtime.h"
//#include "StdAfx.h"
#include "CUDAKernel.h"

__global__ void DoVertexkernel( float4* pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float) width;
	float v = y / (float) height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width+x] = make_float4(u, w, v, __int_as_float(0xff00ff00));
}

CCUDAKernel::CCUDAKernel(void)
{
}

CCUDAKernel::~CCUDAKernel(void)
{
}

void CCUDAKernel::DoVertex(float4* pos, unsigned int width, unsigned int height, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	DoVertexkernel<<<grid, block>>>(pos, width, height, time);
	return ;
}
