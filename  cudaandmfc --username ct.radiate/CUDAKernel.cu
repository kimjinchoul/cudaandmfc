#include "hip/hip_runtime.h"
//#include "StdAfx.h"
#include "CUDAKernel.h"

__global__ void DoVertexKernel( float4* pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float) width;
	float v = y / (float) height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width+x] = make_float4(u, w, v, __int_as_float(0xff00ff00));
}

__global__ void DoTextureKernel(unsigned char* lpCUDATexture, int iTextureWidth, int iTextureHeight, size_t dwTextureBufferPitch, float fTimeStamp)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    float *pixel;
       
    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't 
    // correspond to valid pixels
	if (x >= iTextureWidth || y >= iTextureHeight)
	{
		return;
	}
	
    // get a pointer to the pixel at (x,y)
    pixel = (float *)(lpCUDATexture+y*dwTextureBufferPitch) + 4*x;

	// populate it
	float value_x = 0.5f + 0.5f*cos(fTimeStamp+10.0f*((2.0f*x)/iTextureWidth-1.0f));
	float value_y = 0.5f + 0.5f*cos(fTimeStamp+10.0f*((2.0f*y)/iTextureHeight-1.0f));

	pixel[0] = 0.5*pixel[0] + 0.5*pow(value_x, 3.0f); // red
	pixel[1] = 0.5*pixel[1] + 0.5*pow(value_y, 3.0f); // green
	pixel[2] = 0.5f + 0.5f*cos(fTimeStamp); // blue
	pixel[3] = 1; // alpha

	return;
}

CCUDAKernel::CCUDAKernel(void)
{
}

CCUDAKernel::~CCUDAKernel(void)
{
}

void CCUDAKernel::DoVertex(float4* pos, unsigned int width, unsigned int height, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	DoVertexKernel<<<grid, block>>>(pos, width, height, time);
	return ;
}

void CCUDAKernel::DoTexture(float4 *lpCUDATexture, int iTextureWidth, int iTextureHeight, size_t dwTextureBufferPitch, float fTimeStamp)
{
	dim3 dimBlock(16, 16);
	dim3 dimGrid(iTextureWidth/dimBlock.x, iTextureHeight/dimBlock.y);
    //dim3 dimGrid((iTextureWidth+dimBlock.x-1)/dimBlock.x, (iTextureHeight+dimBlock.y-1)/dimBlock.y);

    DoTextureKernel<<<dimGrid, dimBlock>>>((unsigned char*)lpCUDATexture, iTextureWidth, iTextureHeight, dwTextureBufferPitch, fTimeStamp);

	return;
}